#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "assert.h"
#include "dispatch.h"
#include "image.h"
#include "utils.h"
#include <chrono>
#include "lst.h"

#ifdef USE_CPU_RENDER

int render_image_host(obj_scene_data* h_scene, bvh_t* h_bvh, Vec3* h_img,
                      size_t img_size, render_settings_t settings, char* output_path) {
    printf("Launching host render... \n");
    printf("Rendering %d samples in batches of %d, img size (%d, %d)\n",
           settings.samples, settings.samples_per_round, settings.width, settings.height);

    auto startTime = std::chrono::system_clock::now();

    for (int s = 0; s < settings.samples;) {
        for (int y = 0; y < settings.height; y++) {
            for (int x = 0; x < settings.width; x++) {
                render_host(h_img, h_bvh, h_scene, x, y, settings, s);
            }
        }

        char filename[500];
        sprintf(filename, "render.bmp");
        // sprintf(filename, "render_%.4d.bmp", previous_samples);
        write_bmp(h_img, settings.width, settings.height, filename);

        auto endTime = std::chrono::system_clock::now();
        int elapsedMillis = std::chrono::duration_cast<std::chrono::milliseconds>
            (endTime - startTime).count();
        float elapsedTime = elapsedMillis / (float)1000;

        s += settings.samples_per_round;
        settings.seed++;

        float ksns = settings.width * settings.height * s / (1'000.0 * elapsedTime);

        printf("Rendered %d / %d samples in %.0fs - %.2f samples/s - %.2f kSN/s\n",
               s, settings.samples, elapsedTime, s / elapsedTime, ksns);
    }

    return 0;
}

#else

void render_image_device(scene_t* h_scene, bvh_t* h_bvh, lst_t* h_lst, Vec3* h_img,
                        size_t img_size, settings_t settings, char* output_path) {
    scene_t* d_scene;
    scene_copy_to_device(&d_scene, h_scene);

    bvh_t* d_bvh;
    bvh_copy_device(&d_bvh, h_bvh);

    lst_t* d_lst;
    lst_copy_device(&d_lst, h_lst);

    hipError_t err;

    Vec3* d_img;
    err = hipMalloc(&d_img, img_size);
    if (check_cuda_err(err)) {
        exit(EXIT_FAILURE);
    }

    dim3 threads_per_block(16, 16);  // #threads must be factor of 32 and <= 1024

    int grid_width = (settings.output.width + threads_per_block.x - 1) / threads_per_block.x;
    int grid_height = (settings.output.height + threads_per_block.y - 1) / threads_per_block.y;
    dim3 num_blocks(grid_width, grid_height);

    printf("Launching kernel... \n");
    printf("Rendering %d samples in batches of %d, img size (%d, %d)\n",
           settings.sampling.samples, settings.sampling.samples_per_round, settings.output.width, settings.output.height);
    printf("Kernel params <<<(%u,%u), (%u,%u)>>>\n",
           num_blocks.x, num_blocks.y, threads_per_block.x, threads_per_block.y);
    fflush(stdout);

    auto startTime = std::chrono::system_clock::now();

    for (int s = 0; s < settings.sampling.samples;) {
        render_kernel<<<num_blocks, threads_per_block>>>(d_img, d_bvh, d_scene, d_lst,
                                                         settings, s);
        err = hipDeviceSynchronize();
        if (check_cuda_err(err)) {
            exit(EXIT_FAILURE);
        }

        hipMemcpy(h_img, d_img, img_size, hipMemcpyDeviceToHost);

        write_image(h_img, settings.output.width, settings.output.height, output_path);

        auto endTime = std::chrono::system_clock::now();
        int elapsedMillis = std::chrono::duration_cast<std::chrono::milliseconds>
            (endTime - startTime).count();
        float elapsedTime = elapsedMillis / (float)1000;

        s += settings.sampling.samples_per_round;
        settings.sampling.seed++;

        float megaPixelSamplesS = settings.output.width * settings.output.height * s / (1'000'000.0 * elapsedTime);

        printf("Rendered %d / %d samples in %.1fs - %.2f samples/s - %.2f MPS/s\n",
               s, settings.sampling.samples, elapsedTime, s / elapsedTime, megaPixelSamplesS);
        fflush(stdout);
    }

    err = hipFree(d_img);
    if (check_cuda_err(err)) {
        exit(EXIT_FAILURE);
    }

    free_device_scene(d_scene);
    bvh_free_device(d_bvh);
    lst_free_device(d_lst);
}

#endif