#include "hip/hip_runtime.h"
#include <stdio.h>

#include "assert.h"
#include "bvh.h"
#include "utils.h"
#include <iostream>
#include <limits>

static Vec3
calculate_face_centroid(const scene_t& scene, const face_t& face) {
    float totalArea = 0;
    Vec3 centroid = { 0, 0, 0 };

    for (uint32_t i = 2; i < face.vertexCount; i++) {
        const Vec3& A = scene.vertices[face.vertices[0]].position;
        const Vec3& B = scene.vertices[face.vertices[i - 1]].position;
        const Vec3& C = scene.vertices[face.vertices[i]].position;
        
        Vec3 edge1 = B.cross(A);
        Vec3 edge2 = C.cross(A);
        Vec3 cross = edge1.cross(edge2);
        float triArea = 0.5 * cross.magnitude();
        
        Vec3 triCentroid = (1 / 3.0f) * (A + B + C);

        // weighted sum of centroids
        centroid += triCentroid * triArea;
        totalArea += triArea;
    }

    if (totalArea == 0) {
        return scene.vertices[face.vertices[0]].position;
    }
    centroid /= totalArea;

    return centroid;
}

static void aabb_init(aabb_t& aabb) {
    aabb.min.set(1e30);
    aabb.max.set(-1e30);
}

static float aabb_area(const aabb_t& aabb) {
    Vec3 e = aabb.max - aabb.min;
    return e.x * e.y + e.y * e.z + e.z * e.x;
}

static void aabb_grow_point(aabb_t& aabb, const Vec3& p) {
    aabb.min = Vec3::min(aabb.min, p);
    aabb.max = Vec3::max(aabb.max, p);
}

static void aabb_grow_face(const scene_t& scene, aabb_t& aabb, uint32_t faceIndex) {
    const face_t &face = scene.faces[faceIndex];
    for (uint32_t j = 0; j < face.vertexCount; j++) {
        const Vec3& v = scene.vertices[face.vertices[j]].position;
        aabb_grow_point(aabb, v);
    }
}

static void update_node_bounds(const scene_t& scene, bvh_t& bvh, uint32_t nodeIndex) {
    bvh_node_t& node = bvh.nodes[nodeIndex];
    aabb_init(node.bounds);
    for (uint32_t i = node.start; i < node.end; i++) {
        aabb_grow_face(scene, node.bounds, bvh.indices[i]);
    }
}

void swap(uint32_t* a, uint32_t* b) {
    uint32_t t = *a;
    *a = *b;
    *b = t;
}

static float
evaluate_sah(const scene_t& scene,
             bvh_t& bvh, bvh_node_t& node, int axis, float splitPos) {
    // determine triangle counts and bounds for this split candidate
    aabb_t leftBox, rightBox;
    aabb_init(leftBox);
    aabb_init(rightBox);
    uint32_t leftCount = 0, rightCount = 0;
    for (uint32_t i = node.start; i < node.end; i++) {
        float centroid = bvh.centroids[bvh.indices[i]][axis];
        if (centroid < splitPos) {
            leftCount++;
            aabb_grow_face(scene, leftBox, bvh.indices[i]);
        } else {
            rightCount++;
            aabb_grow_face(scene, rightBox, bvh.indices[i]);
        }
    }

    float cost = leftCount * aabb_area(leftBox) + rightCount * aabb_area(rightBox);
    return cost > 0 ? cost : 1e30;
}

// static float
// find_split_plane_centroids(const scene_t& scene, bvh_t& bvh, bvh_node_t& node,
//                            int* bestAxis, float* bestSplit) {
//     float bestCost = 1e30;
//     for (int a = 0; a < 3; a++) {
//         for (uint32_t i = node.start; i < node.end; i++) {
//             float candidate = bvh.centroids[bvh.indices[i]][a];
//             float cost = evaluate_sah(scene, bvh, node, a, candidate);
//             if (cost < bestCost) {
//                 *bestAxis = a;
//                 *bestSplit = candidate;
//                 bestCost = cost;
//             }
//         }
//     }
//     assert(*bestAxis >= 0);
//     return bestCost;
// }

static float
find_best_split_plane_bands(const scene_t& scene, bvh_t& bvh, bvh_node_t& node,
                      int* bestAxis, float* bestSplit, int numBands) {

    AABB centroidBounds;
    aabb_init(centroidBounds);
    for (size_t i = node.start; i < node.end; i++) {
        const Vec3& c = bvh.centroids[bvh.indices[i]];
        centroidBounds = centroidBounds.including(c);
    }

    float bestCost = 1e30;
    for (int a = 0; a < 3; a++) {
        float boundsMin = centroidBounds.min[a];
        float boundsMax = centroidBounds.max[a];
        if (std::abs(boundsMin - boundsMax) < std::numeric_limits<float>::epsilon()) {
            continue;
        }
        float scale = (boundsMax - boundsMin) / numBands;
        for (int i = 1; i < numBands; i++) {
            float candidate = boundsMin + i * scale;
            float cost = evaluate_sah(scene, bvh, node, a, candidate);
            if (cost < bestCost) {
                *bestAxis = a;
                *bestSplit = candidate;
                bestCost = cost;
            }
        }
    }
    assert(*bestAxis >= 0);
    return bestCost;
}

#define N_BANDS 8

static float
find_best_split_plane(const scene_t& scene, bvh_t& bvh, bvh_node_t& node,
                      int* bestAxis, float* bestSplit) {
    // int count = node.end - node.start;
    // if (count < N_BANDS) {
    //     return find_split_plane_centroids(scene, bvh, node, bestAxis, bestSplit);
    // } else {
    return find_best_split_plane_bands(scene, bvh, node, bestAxis, bestSplit, N_BANDS);
    // }
}

static void
subdivide(const scene_t& scene, bvh_t& bvh, uint32_t nodeIndex, bvh_stats_t& stats, int depth) {

    stats.maxDepth = std::max(stats.maxDepth, depth);

    bvh_node_t& node = bvh.nodes[nodeIndex];
    int count = node.end - node.start;
    if (count <= 2) {
        return;  // stop criterion
    }
    // uint32_t i = split_by_average(bvh, node, count, stats);
    // uint32_t i = split_by_median(bvh, node, count, stats);

    int bestAxis = -1;
    float bestPos = 0;
    float bestCost = find_best_split_plane(scene, bvh, node, &bestAxis, &bestPos);

    // partitioning
    int i = node.start;
    int j = node.end - 1;
    while (i <= j) {
        if (bvh.centroids[bvh.indices[i]][bestAxis] < bestPos) {
            i++;
        } else {
            swap(&bvh.indices[i], &bvh.indices[j--]);
        }
    }

    int leftCount = i - node.start;
    if (leftCount == 0 || leftCount == count) {
        // printf("Skipped split\n");
        stats.totalSkippedFaces += count;
        return;
    }

    uint32_t leftIndex = bvh.nodeCount++;
    uint32_t rightIndex = bvh.nodeCount++;
    bvh_node_t& left = bvh.nodes[leftIndex];
    bvh_node_t& right = bvh.nodes[rightIndex];
    left.start = node.start;
    left.end = right.start = i;
    right.end = node.end;

    // printf("Split: %u => %u / %u\n", left->start, left->end - left->start, right->end - right->start);

    time_t currTime;
    time(&currTime);
    double elapsed_seconds = difftime(currTime, stats.lastInfo);
    if (elapsed_seconds > 1.0) {
        printf("created %u bvh nodes (of at most %d)\n", bvh.nodeCount, bvh.maxNodeCount);
        stats.lastInfo = currTime;
    }

    node.leftChild = leftIndex;
    node.rightChild = rightIndex;
    node.start = node.end = 0;  // make non-leaf
    update_node_bounds(scene, bvh, leftIndex);
    update_node_bounds(scene, bvh, rightIndex);
    subdivide(scene, bvh, leftIndex, stats, depth + 1);
    subdivide(scene, bvh, rightIndex, stats, depth + 1);
}

static void
calculate_stats(const scene_t& scene, bvh_t& bvh, uint32_t nodeIndex, bvh_stats_t& stats) {
    stats.numberLeaves = 0;
    stats.averageLeafSize = 0;
    for (uint32_t i = 0; i < bvh.nodeCount; i++) {
        bvh_node_t& node = bvh.nodes[i];
        uint32_t faces = node.end - node.start;
        if (faces > 0) {
            stats.numberLeaves++;
            stats.averageLeafSize += faces;
        }
    }
    stats.averageLeafSize /= stats.numberLeaves;
}

static void
print_stats(const scene_t& scene, bvh_t& bvh, uint32_t nodeIndex, bvh_stats_t& stats) {
    if (!doVerbosePrinting) {
        return;
    }

    printf("\nbvh_t stats:\n");
    printf("  node count = %u\n", bvh.nodeCount);
    printf("  optimal node count = %u\n", bvh.maxNodeCount);
    printf("  number leafs = %u\n", stats.numberLeaves);
    printf("  skipped faces = %u / %u = %.3f\n",
           stats.totalSkippedFaces, bvh.primitiveCount,
           stats.totalSkippedFaces / (float)bvh.primitiveCount);
    printf("  average leaf size = %.2f\n", stats.averageLeafSize);
    printf("  max tree height =  %d\n", stats.maxDepth);
}

void
bvh_build(bvh_t& bvh, const scene_t& scene) {
    printf("Building bvh_t...  \n");
    fflush(stdout);

    bvh_stats_t stats;
    memset(&stats, 0, sizeof(bvh_stats_t));
    time(&stats.lastInfo);

    bvh.primitiveCount = scene.faces.count;
    bvh.maxNodeCount = 2 * bvh.primitiveCount - 1;

    bvh.nodeCount = 0;
    bvh.nodes = (bvh_node_t*)malloc(sizeof(bvh_node_t) * bvh.maxNodeCount);
    assert(bvh.nodes && "malloc");

    // mutable primitive list for sorting faces
    bvh.indices = (uint32_t*)malloc(sizeof(uint32_t) * bvh.primitiveCount);
    assert(bvh.indices && "malloc");

    for (uint32_t i = 0; i < bvh.primitiveCount; i++) {
        bvh.indices[i] = i;
    }
    // calculate centroids, accesses work with scene face indices
    bvh.centroids = (Vec3*)malloc(sizeof(Vec3) * bvh.primitiveCount);
    assert(bvh.centroids && "malloc");

    for (uint32_t i = 0; i < bvh.primitiveCount; i++) {
        bvh.centroids[i] = calculate_face_centroid(scene, scene.faces[i]);
    }

    uint32_t rootIndex = bvh.nodeCount++;
    bvh_node_t& root = bvh.nodes[rootIndex];
    root.start = 0;
    root.end = bvh.primitiveCount;
    update_node_bounds(scene, bvh, rootIndex);
    subdivide(scene, bvh, rootIndex, stats, 1);

    calculate_stats(scene, bvh, rootIndex, stats);
    print_stats(scene, bvh, rootIndex, stats);

    // not needed anymore
    free(bvh.centroids);
    bvh.centroids = NULL;

    if (stats.maxDepth + 1 > BVH_TRAVERSAL_STACK_SIZE) {
        printf("ERROR bvh max height (%d) is too large, increase BVH_TRAVERSAL_STACK_SIZE (%d)\n", 
            stats.maxDepth, BVH_TRAVERSAL_STACK_SIZE);
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
}

void bvh_free_host(bvh_t& h_bvh) {
    free(h_bvh.nodes);
    h_bvh.nodes = NULL;
    free(h_bvh.indices);
    h_bvh.indices = NULL;
}

void bvh_copy_device(bvh_t** d_bvh, const bvh_t* h_bvh) {
    printf("Copying bvh_t to device... ");

    bvh_t m_bvh = *h_bvh;

    hipError_t err;
    size_t curr_bytes, total_bytes;
    total_bytes = 0;

    curr_bytes = sizeof(bvh_node_t) * m_bvh.maxNodeCount;
    total_bytes += curr_bytes;
    err = hipMalloc(&m_bvh.nodes, curr_bytes);
    if (check_cuda_err(err)) exit(EXIT_FAILURE);
    err = hipMemcpy(m_bvh.nodes, h_bvh->nodes,
                     curr_bytes, hipMemcpyHostToDevice);
    if (check_cuda_err(err)) exit(EXIT_FAILURE);

    curr_bytes = sizeof(uint32_t) * m_bvh.primitiveCount;
    total_bytes += curr_bytes;
    err = hipMalloc(&m_bvh.indices, curr_bytes);
    if (check_cuda_err(err)) exit(EXIT_FAILURE);
    err = hipMemcpy(m_bvh.indices, h_bvh->indices,
                     curr_bytes, hipMemcpyHostToDevice);
    if (check_cuda_err(err)) exit(EXIT_FAILURE);

    total_bytes += sizeof(bvh_t);
    err = hipMalloc(d_bvh, sizeof(bvh_t));
    if (check_cuda_err(err)) exit(EXIT_FAILURE);
    err = hipMemcpy(*d_bvh, &m_bvh,
                     sizeof(bvh_t), hipMemcpyHostToDevice);
    if (check_cuda_err(err)) exit(EXIT_FAILURE);

    printf("Done [%ldkB]\n", total_bytes / 1000);
}

void bvh_free_device(bvh_t* d_bvh) {
    hipError_t err;
    bvh_t m_bvh;
    err = hipMemcpy(&m_bvh, d_bvh, sizeof(bvh_t), hipMemcpyDeviceToHost);
    if (check_cuda_err(err)) exit(EXIT_FAILURE);
    err = hipFree(d_bvh);
    if (check_cuda_err(err)) exit(EXIT_FAILURE);
    err = hipFree(m_bvh.nodes);
    if (check_cuda_err(err)) exit(EXIT_FAILURE);
    err = hipFree(m_bvh.indices);
    if (check_cuda_err(err)) exit(EXIT_FAILURE);
}

#define RAY_NO_HIT 1e30f

static PLATFORM float
intersect_aabb(const aabb_t& aabb, const Ray& ray, float min_t) {
    Vec3 t1 = (aabb.min - ray.o) / ray.r;
    Vec3 t2 = (aabb.max - ray.o) / ray.r;
    Vec3 tminv = Vec3::min(t1, t2);
    Vec3 tmaxv = Vec3::max(t1, t2);

    float tmin = tminv.maxComponent();
    float tmax = tmaxv.minComponent();

    if (tmax >= tmin && tmin < min_t && tmax > 0) {
        return tmin;
    } else {
        return RAY_NO_HIT;
    }
}

PLATFORM void
bvh_intersect(const __restrict__ bvh_t* bvh, uint32_t nodeIndex,
              const __restrict__ scene_t* scene, const Ray& ray, intersection_t& hit) {
    bvh_node_t &node = bvh->nodes[nodeIndex];
    if (!intersect_aabb(node.bounds, ray, hit.distance)) {
        return;
    }
    if (node.end - node.start > 0) {
        // is leaf
        for (uint32_t i = node.start; i < node.end; i++) {
            intersect_face(scene, ray, hit, bvh->indices[i]);
        }
    } else {
        bvh_intersect(bvh, node.leftChild, scene, ray, hit);
        bvh_intersect(bvh, node.rightChild, scene, ray, hit);
    }
}

PLATFORM void
bvh_intersect_iterative(const __restrict__ bvh_t* bvh,
                        const __restrict__ scene_t* scene, const Ray& ray, intersection_t& hit) {

    bvh_node_t* stack[BVH_TRAVERSAL_STACK_SIZE];
    int depth = 0;
    stack[0] = &bvh->nodes[0];

    while (depth >= 0) {
        bvh_node_t* node = stack[depth--];

        while (node != NULL) {

            if (node->end - node->start > 0) {
                // is leaf
                for (uint32_t i = node->start; i < node->end; i++) {
                    intersect_face(scene, ray, hit, bvh->indices[i]);
                }
                node = NULL;
            } else {

                bvh_node_t* child1 = &bvh->nodes[node->leftChild];
                bvh_node_t* child2 = &bvh->nodes[node->rightChild];

                float t1 = intersect_aabb(child1->bounds, ray, hit.distance);
                float t2 = intersect_aabb(child2->bounds, ray, hit.distance);

                if (t2 < t1) {
                    bvh_node_t* tempc = child1;
                    child1 = child2;
                    child2 = tempc;
                    float tempt = t1;
                    t1 = t2;
                    t2 = tempt;
                }

                if (t1 == RAY_NO_HIT) {
                    node = NULL;
                } else {
                    node = child1;
                    if (t2 != RAY_NO_HIT) {
                        stack[++depth] = child2;
                    }
                }
            }
        }
    }
}

