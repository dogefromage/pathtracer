#include "hip/hip_runtime.h"
#include <stdio.h>

#include "image.h"
#include <cstdint>

uint8_t clamp_channel(float c) {
    if (c < 0.0) c = 0.0;
    if (c > 1.0) c = 1.0;
    return (uint8_t)(255 * c);
}

Vec3 linear_to_gamma(Vec3 c) {
    return Vec3(
        sqrtf(c.x),
        sqrtf(c.y),
        sqrtf(c.z)
    );
}

Vec3 color_correct(Vec3 c) {
    return linear_to_gamma(c);
}

void write_bmp(Vec3* pixels, int width, int height, const char* filename) {
    unsigned int header[14];
    int i, j;
    FILE* fp = fopen(filename, "wb");
    uint8_t pad[3] = {0, 0, 0};

    header[0] = 0x4d420000;
    header[1] = 54 + 3 * height * width;
    header[2] = 0;
    header[3] = 54;
    header[4] = 40;
    header[5] = width;
    header[6] = height;
    header[7] = 0x00180001;
    header[8] = 0;
    header[9] = 3 * width * height;
    header[10] = header[11] = header[12] = header[13] = 0;

    fwrite((uint8_t*)header + 2, 1, 54, fp);
    fflush(fp);

    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            int pixel = i * width + j;
            Vec3 c = pixels[pixel];
            Vec3 c_corr = color_correct(c);
            uint8_t R = clamp_channel(c_corr.x);
            uint8_t G = clamp_channel(c_corr.y);
            uint8_t B = clamp_channel(c_corr.z);
            fwrite(&B, 1, 1, fp);
            fwrite(&G, 1, 1, fp);
            fwrite(&R, 1, 1, fp);
        }
        fwrite(pad, width % 4, 1, fp);
    }

    fclose(fp);
}